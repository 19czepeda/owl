#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>


// ==================================================================
// bounding box programs - since these don't actually use the material
// they're all the same irrespective of geometry type, so use a
// template ...
// ==================================================================
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
  const Sphere sphere = self.prims[primID].sphere;
  primBounds = box3f()
    .extend(sphere.center - sphere.radius)
    .extend(sphere.center + sphere.radius);
}

OPTIX_BOUNDS_PROGRAM(MetalSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<MetalSpheresGeom>(geomData,primBounds,primID); }

OPTIX_BOUNDS_PROGRAM(LambertianSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<LambertianSpheresGeom>(geomData,primBounds,primID); }

OPTIX_BOUNDS_PROGRAM(DielectricSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<DielectricSpheresGeom>(geomData,primBounds,primID); }


// ==================================================================
// intersect programs - still all the same, since they don't use the
// material, either
// ==================================================================

template<typename SpheresGeomType>
inline __device__ void intersectProg()
{
  const int primID = optixGetPrimitiveIndex();
  const auto &self
    = owl::getProgramData<SpheresGeomType>().prims[primID];
  
  const vec3f org  = optixGetWorldRayOrigin();
  const vec3f dir  = optixGetWorldRayDirection();
  float hit_t      = optixGetRayTmax();
  const float tmin = optixGetRayTmin();

  const vec3f oc = org - self.sphere.center;
  const float a = dot(dir,dir);
  const float b = dot(oc, dir);
  const float c = dot(oc, oc) - self.sphere.radius * self.sphere.radius;
  const float discriminant = b * b - a * c;
  
  if (discriminant < 0.f) return;

  {
    float temp = (-b - sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
      
  {
    float temp = (-b + sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
  if (hit_t < optixGetRayTmax()) {
    optixReportIntersection(hit_t, 0);
  }
}


OPTIX_INTERSECT_PROGRAM(MetalSpheres)()
{ intersectProg<MetalSpheresGeom>(); }

OPTIX_INTERSECT_PROGRAM(LambertianSpheres)()
{ intersectProg<LambertianSpheresGeom>(); }

OPTIX_INTERSECT_PROGRAM(DielectricSpheres)()
{ intersectProg<DielectricSpheresGeom>(); }


// ==================================================================
// plumbing for closest hit
// ==================================================================

OPTIX_CLOSEST_HIT_PROGRAM(MetalSpheres)()
{
  const int primID = optixGetPrimitiveIndex();
  const MetalSphere &self
    = owl::getProgramData<MetalSpheresGeom>().prims[primID];
  
  vec3f &prd = owl::getPRD<vec3f>();

  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f Ng    = normalize(hit_P-self.sphere.center);

  prd = (.2f + .8f*fabs(dot(dir,Ng)))*self.metal.albedo;
}

OPTIX_CLOSEST_HIT_PROGRAM(LambertianSpheres)()
{
  const int primID = optixGetPrimitiveIndex();
  const LambertianSphere &self
    = owl::getProgramData<LambertianSpheresGeom>().prims[primID];
  
  vec3f &prd = owl::getPRD<vec3f>();

  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f Ng    = normalize(hit_P-self.sphere.center);

  prd = (.2f + .8f*fabs(dot(dir,Ng)))*self.lambertian.albedo;
}


OPTIX_CLOSEST_HIT_PROGRAM(DielectricSpheres)()
{
  const int primID = optixGetPrimitiveIndex();
  const DielectricSphere &self
    = owl::getProgramData<DielectricSpheresGeom>().prims[primID];
  
  vec3f &prd = owl::getPRD<vec3f>();

  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f Ng    = normalize(hit_P-self.sphere.center);

  prd = (.2f + .8f*fabs(dot(dir,Ng)));//*self.dielectric.albedo;
}








// ==================================================================
// miss and raygen
// ==================================================================

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();

  const vec3f unit_direction = normalize((vec3f)optixGetWorldRayDirection());
  const float t = 0.5f*(unit_direction.y + 1.0f);
  const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  vec3f &prd = owl::getPRD<vec3f>();
  prd = c;
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  
  if (pixelID.x >= self.fbSize.x) return;
  if (pixelID.y >= self.fbSize.y) return;

  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;

  // const vec3f rd = camera_lens_radius * random_in_unit_disk(rnd);
  // const vec3f lens_offset = camera_u * rd.x + camera_v * rd.y;
  const vec3f origin = self.camera.origin // + lens_offset
    ;
  const vec3f direction
    = self.camera.lower_left_corner
    + screen.u * self.camera.horizontal
    + screen.v * self.camera.vertical
    - self.camera.origin;
  
  ray.origin = origin;
  ray.direction = direction;

  vec3f color;
  owl::trace(/*accel to trace against*/self.world,
             /*the ray to trace*/ ray,
             /*numRayTypes*/1,
             /*prd*/color);
    
  const int fbOfs = pixelID.x+self.fbSize.x*(self.fbSize.y-1-pixelID.y);
  self.fbPtr[fbOfs]
    = owl::make_rgba(color);
}


