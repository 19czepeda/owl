#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

#define NUM_SAMPLES_PER_PIXEL 128

inline __device__
float schlick(float cosine,
              float ref_idx)
{
  float r0 = (1.0f - ref_idx) / (1.0f + ref_idx);
  r0 = r0 * r0;
  return r0 + (1.0f - r0)*powf((1.0f - cosine), 5.0f);
}

inline __device__
bool refract(const vec3f& v,
             const vec3f& n,
             float ni_over_nt,
             vec3f &refracted)
{
  vec3f uv = normalize(v);
  float dt = dot(uv, n);
  float discriminant = 1.0f - ni_over_nt * ni_over_nt*(1 - dt * dt);
  if (discriminant > 0.f) {
    refracted = ni_over_nt * (uv - n * dt) - n * sqrtf(discriminant);
    return true;
  }
  else
    return false;
}

inline __device__
vec3f reflect(const vec3f &v,
              const vec3f &n)
{
  return v - 2.0f*dot(v, n)*n;
}

struct DRand48
{
  /*! initialize the random number generator with a new seed (usually
      per pixel) */
  inline __device__ void init(int seed = 0)
  {
    state = seed;
    for (int warmUp=0;warmUp<10;warmUp++)
      (*this)();
  }

  /*! get the next 'random' number in the sequence */
  inline __device__ float operator() ()
  {
    const uint64_t a = 0x5DEECE66DULL;
    const uint64_t c = 0xBULL;
    const uint64_t mask = 0xFFFFFFFFFFFFULL;
    state = a*state + c;
    return float((state & mask) / float(mask+1ULL));
  }

  uint64_t state;
};



typedef enum {
  /*! ray could get properly bounced, and is still alive */
  rayGotBounced,
  /*! ray could not get scattered, and should get cancelled */
  rayGotCancelled,
  /*! ray didn't hit anything, and went into the environemnt */
  rayDidntHitAnything
} ScatterEvent;

/*! "per ray data" (PRD) for our sample's rays. In the simple example, there is only
  one ray type, and it only ever returns one thing, which is a color (everything else
  is handled through the recursion). In addition to that return type, rays have to
  carry recursion state, which in this case are recursion depth and random number state */
struct PerRayData {
  DRand48 random;
  struct {
    ScatterEvent scatterEvent;
    vec3f        scattered_origin;
    vec3f        scattered_direction;
    vec3f        attenuation;
  } out;
};




inline __device__ vec3f randomPointOnUnitDisc(DRand48 &random) {
  vec3f p;
  do {
    p = 2.0f*vec3f(random(), random(), 0.f) - vec3f(1.f, 1.f, 0.f);
  } while (dot(p, p) >= 1.0f);
  return p;
}


#define RANDVEC3F vec3f(rnd(),rnd(),rnd())

inline __device__ vec3f randomPointInUnitSphere(DRand48 &rnd) {
  vec3f p;
  do {
    p = 2.0f*RANDVEC3F - vec3f(1, 1, 1);
  } while (dot(p,p) >= 1.0f);
  return p;
}


inline __device__
bool scatter(const Lambertian &lambertian,
             const vec3f &P,
             vec3f N,
             // const owl::Ray &ray_in,
             PerRayData &prd)
{
  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();

  if (dot(N,dir)  > 0.f)
    N = -N;
  N = normalize(N);

  const vec3f target
    = P + (N + randomPointInUnitSphere(prd.random));

  
  // return scattering event
  prd.out.scattered_origin    = P;
  prd.out.scattered_direction = (target-P);
  prd.out.attenuation         = lambertian.albedo;
  return true;
}

inline __device__
bool scatter(const Dielectric &dielectric,
             const vec3f &P,
             vec3f N,
             PerRayData &prd)
{
  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = normalize((vec3f)optixGetWorldRayDirection());

  N = normalize(N);
  vec3f outward_normal;
  vec3f reflected = reflect(dir,N);
  float ni_over_nt;
  prd.out.attenuation = vec3f(1.f, 1.f, 1.f); 
  vec3f refracted;
  float reflect_prob;
  float cosine;
  
  if (dot(dir,N) > 0.f) {
    outward_normal = -N;
    ni_over_nt = dielectric.ref_idx;
    cosine = dot(dir, N);// / vec3f(dir).length();
    cosine = sqrtf(1.f - dielectric.ref_idx*dielectric.ref_idx*(1.f-cosine*cosine));
  }
  else {
    outward_normal = N;
    ni_over_nt = 1.0 / dielectric.ref_idx;
    cosine = -dot(dir, N);// / vec3f(dir).length();
  }
  if (refract(dir, outward_normal, ni_over_nt, refracted)) 
    reflect_prob = schlick(cosine, dielectric.ref_idx);
  else 
    reflect_prob = 1.f;

  prd.out.scattered_origin = P;
  if (prd.random() < reflect_prob) 
    prd.out.scattered_direction = reflected;
  else 
    prd.out.scattered_direction = refracted;
  
  return true;
}

inline __device__
bool scatter(const Metal &metal,
             const vec3f &P,
             vec3f N,
             PerRayData &prd)
{
  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();

  if (dot(N,dir)  > 0.f)
    N = -N;
  N = normalize(N);
  
  vec3f reflected = reflect(normalize(dir),N);
  prd.out.scattered_origin    = P;
  prd.out.scattered_direction
    = (reflected+metal.fuzz*randomPointInUnitSphere(prd.random));
  prd.out.attenuation         = metal.albedo;
  return (dot(prd.out.scattered_direction, N) > 0.f);
}



// ==================================================================
// bounding box programs - since these don't actually use the material
// they're all the same irrespective of geometry type, so use a
// template ...
// ==================================================================
template<typename SphereGeomType>
inline __device__ void boundsProg(const void *geomData,
                                  box3f &primBounds,
                                  const int primID)
{
  const SphereGeomType &self = *(const SphereGeomType*)geomData;
  const Sphere sphere = self.prims[primID].sphere;
  primBounds = box3f()
    .extend(sphere.center - sphere.radius)
    .extend(sphere.center + sphere.radius);
}

OPTIX_BOUNDS_PROGRAM(MetalSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<MetalSpheresGeom>(geomData,primBounds,primID); }

OPTIX_BOUNDS_PROGRAM(LambertianSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<LambertianSpheresGeom>(geomData,primBounds,primID); }

OPTIX_BOUNDS_PROGRAM(DielectricSpheres)(const void  *geomData,
                                        box3f       &primBounds,
                                        const int    primID)
{ boundsProg<DielectricSpheresGeom>(geomData,primBounds,primID); }


// ==================================================================
// intersect programs - still all the same, since they don't use the
// material, either
// ==================================================================

template<typename SpheresGeomType>
inline __device__ void intersectProg()
{
  const int primID = optixGetPrimitiveIndex();
  const auto &self
    = owl::getProgramData<SpheresGeomType>().prims[primID];
  
  const vec3f org  = optixGetWorldRayOrigin();
  const vec3f dir  = optixGetWorldRayDirection();
  float hit_t      = optixGetRayTmax();
  const float tmin = optixGetRayTmin();

  const vec3f oc = org - self.sphere.center;
  const float a = dot(dir,dir);
  const float b = dot(oc, dir);
  const float c = dot(oc, oc) - self.sphere.radius * self.sphere.radius;
  const float discriminant = b * b - a * c;
  
  if (discriminant < 0.f) return;

  {
    float temp = (-b - sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
      
  {
    float temp = (-b + sqrtf(discriminant)) / a;
    if (temp < hit_t && temp > tmin) 
      hit_t = temp;
  }
  if (hit_t < optixGetRayTmax()) {
    optixReportIntersection(hit_t, 0);
  }
}


OPTIX_INTERSECT_PROGRAM(MetalSpheres)()
{ intersectProg<MetalSpheresGeom>(); }

OPTIX_INTERSECT_PROGRAM(LambertianSpheres)()
{ intersectProg<LambertianSpheresGeom>(); }

OPTIX_INTERSECT_PROGRAM(DielectricSpheres)()
{ intersectProg<DielectricSpheresGeom>(); }


// ==================================================================
// plumbing for closest hit
// ==================================================================

template<typename SpheresGeomType>
inline __device__
void closestHit()
{
  const int primID = optixGetPrimitiveIndex();
  const auto &self
    = owl::getProgramData<SpheresGeomType>().prims[primID];
  
  PerRayData &prd = owl::getPRD<PerRayData>();

  const vec3f org   = optixGetWorldRayOrigin();
  const vec3f dir   = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f N     = (hit_P-self.sphere.center);

  prd.out.scatterEvent
    = scatter(self.material,
              hit_P,N,//ray,
              prd)
    ? rayGotBounced
    : rayGotCancelled;
}

OPTIX_CLOSEST_HIT_PROGRAM(MetalSpheres)()
{ closestHit<MetalSpheresGeom>(); }
OPTIX_CLOSEST_HIT_PROGRAM(LambertianSpheres)()
{ closestHit<LambertianSpheresGeom>(); }
OPTIX_CLOSEST_HIT_PROGRAM(DielectricSpheres)()
{ closestHit<DielectricSpheresGeom>(); }









// ==================================================================
// miss and raygen
// ==================================================================

inline __device__
vec3f missColor(const Ray &ray)
{
  const vec2i pixelID = owl::getLaunchIndex();

  const vec3f rayDir = normalize(ray.direction);
  const float t = 0.5f*(rayDir.y + 1.0f);
  const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
  PerRayData &prd = owl::getPRD<PerRayData>();
  prd.out.scatterEvent = rayDidntHitAnything;
  // const vec2i pixelID = owl::getLaunchIndex();

  // const MissProgData &self = owl::getProgramData<MissProgData>();

  // const vec3f unit_direction = normalize((vec3f)optixGetWorldRayDirection());
  // const float t = 0.5f*(unit_direction.y + 1.0f);
  // const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  // vec3f &prd = owl::getPRD<vec3f>();
  // prd = c;
}



inline __device__
vec3f tracePath(const RayGenData &self,
                owl::Ray &ray, PerRayData &prd)
{
  vec3f attenuation = 1.f;
  
  /* iterative version of recursion, up to depth 50 */
  for (int depth=0;depth<50;depth++) {
    owl::trace(/*accel to trace against*/self.world,
               /*the ray to trace*/ ray,
               /*numRayTypes*/1,
               /*prd*/prd);
    
    if (prd.out.scatterEvent == rayDidntHitAnything)
      /* ray got 'lost' to the environment - 'light' it with miss
         shader */
      return attenuation * missColor(ray);
    else if (prd.out.scatterEvent == rayGotCancelled)
      return vec3f(0.f);

    else { // ray is still alive, and got properly bounced
      attenuation *= prd.out.attenuation;
      ray = owl::Ray(/* origin   : */ prd.out.scattered_origin,
                     /* direction: */ prd.out.scattered_direction,
                     /* ray type : */ 0,
                     /* tmin     : */ 1e-3f,
                     /* tmax     : */ 1e10f);
    }
  }
  // recursion did not terminate - cancel it
  return vec3f(0.f);
}

OPTIX_RAYGEN_PROGRAM(rayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  
  if (pixelID.x >= self.fbSize.x) return;
  if (pixelID.y >= self.fbSize.y) return;
  const int pixelIdx = pixelID.x+self.fbSize.x*(self.fbSize.y-1-pixelID.y);

  // for multi-gpu: only render every deviceCount'th column of 32 pixels:
  if (((pixelID.x/32) % self.deviceCount) != self.deviceIndex)
    return;
  
  PerRayData prd;
  prd.random.init(pixelIdx);
  
  vec3f color = 0.f;
  for (int sampleID=0;sampleID<NUM_SAMPLES_PER_PIXEL;sampleID++) {
    owl::Ray ray;
    
    const vec2f pixelSample(prd.random(),prd.random());
    const vec2f screen
      = (vec2f(pixelID)+pixelSample)
      / vec2f(self.fbSize);
    const vec3f origin = self.camera.origin // + lens_offset
      ;
    const vec3f direction
      = self.camera.lower_left_corner
      + screen.u * self.camera.horizontal
      + screen.v * self.camera.vertical
      - self.camera.origin;
  
    ray.origin = origin;
    ray.direction = direction;

    color += tracePath(self, ray, prd);
  }
    
  self.fbPtr[pixelIdx]
    = owl::make_rgba(color * (1.f / NUM_SAMPLES_PER_PIXEL));
}


