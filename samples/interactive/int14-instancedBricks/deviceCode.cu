#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;
  ray.origin    
    = self.camera.pos;
  ray.direction 
    = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);

  vec3f color;
  owl::traceRay(/*accel to trace against*/self.world,
                /*the ray to trace*/ray,
                /*prd*/color);
    
  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
  self.fbPtr[fbOfs]
    = owl::make_rgba(color);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Ng     = normalize(cross(B-A,C-A));

  const unsigned int instanceID = optixGetInstanceId();
  const vec3f color = self.colorPerInstance[instanceID];

  const vec3f rayDir = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*color;
}

inline __device__ box3f boxIndicesToBounds(vec3f anchor, float worldScale, int x, int y, int z)
{
    const vec3f boxmin = anchor + vec3f( x, y, z ) * 2.0f*worldScale;
    const vec3f boxmax = boxmin + 2.0f*worldScale;
    return box3f(boxmin, boxmax);
} 

OPTIX_BOUNDS_PROGRAM(VoxGeom)(const void *geomData,
                              box3f &primBounds,
                              const int primID)
{
  const VoxGeomData &self = *(const VoxGeomData*)geomData;
  uchar4 indices = self.prims[primID];
  primBounds = boxIndicesToBounds(self.anchor, self.worldScale, indices.x, indices.y, indices.z);
}

OPTIX_INTERSECT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  const box3f primBounds = boxIndicesToBounds(self.anchor, self.worldScale, indices.x, indices.y, indices.z);

  const vec3f org  = optixGetObjectRayOrigin();
  const vec3f dir  = optixGetObjectRayDirection();
  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  vec3f t0 = (primBounds.lower - org) / dir;
  vec3f t1 = (primBounds.upper - org) / dir;
  vec3f tnear = owl::min(t0, t1);
  vec3f tfar = owl::max(t0, t1);
  float tmin = reduce_max(tnear);
  float tmax = reduce_min(tfar);

  if (tmin <= tmax) {

        bool check_second = true;
        if (tmin > ray_tmin && tmin < ray_tmax) {
            if (optixReportIntersection( tmin, 0)) { 
                check_second = false;
            }
        } 
        if (check_second && tmax > ray_tmin && tmax < ray_tmax) {
            // ray might have started inside the box.
            // Can remove this case if camera is guaranteed to be outside
            optixReportIntersection( tmax, 0);
        }
    }
}

static __device__ vec3f makeBoxNormal( float3 t0, float3 t1, float t)
{
    if (t == t1.x) 
        return vec3f(1.0f,  0.0f,  0.0f);
    else if (t == t0.x)
        return vec3f( -1.0f,  0.0f,  0.0f);
    else if (t == t1.y)
        return vec3f( 0.0f,  1.0f,  0.0f);
    else if (t == t0.y)
        return vec3f( 0.0f, -1.0f,  0.0f);
    else if (t == t1.z)
        return vec3f( 0.0f,  0.0f,  1.0f);
    else //if (t == t0.z)
        return vec3f( 0.0f,  0.0f, -1.0f);
}

OPTIX_CLOSEST_HIT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  const box3f primBounds = boxIndicesToBounds(self.anchor, self.worldScale, indices.x, indices.y, indices.z);

  vec3f &prd = owl::getPRD<vec3f>();
  const vec3f org = optixGetWorldRayOrigin();
  const vec3f dir = optixGetWorldRayDirection();
  const float hit_t = optixGetRayTmax();

  // Find normal for whichever face we hit
  vec3f t0 = (primBounds.lower - org) / dir;
  vec3f t1 = (primBounds.upper - org) / dir;
  vec3f Ng = makeBoxNormal(t0, t1, hit_t);

  // Convert 8 bit color to float
  const int ci = self.prims[primID].w;
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  prd = (.2f + .8f*fabs(dot(dir,Ng)))*color;


}

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = (pattern&1) ? self.color1 : self.color0;
}

