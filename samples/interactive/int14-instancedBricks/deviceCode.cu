#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();

  const vec2f screen = (vec2f(pixelID)+vec2f(.5f)) / vec2f(self.fbSize);
  owl::Ray ray;
  ray.origin    
    = self.camera.pos;
  ray.direction 
    = normalize(self.camera.dir_00
                + screen.u * self.camera.dir_du
                + screen.v * self.camera.dir_dv);

  vec3f color;
  owl::traceRay(/*accel to trace against*/self.world,
                /*the ray to trace*/ray,
                /*prd*/color);
    
  const int fbOfs = pixelID.x+self.fbSize.x*pixelID.y;
  self.fbPtr[fbOfs]
    = owl::make_rgba(color);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  vec3f &prd = owl::getPRD<vec3f>();

  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Ng     = normalize(cross(B-A,C-A));

  const unsigned int instanceID = optixGetInstanceId();
  const vec3f color = self.colorPerInstance[instanceID];

  const vec3f rayDir = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*color;
}

OPTIX_BOUNDS_PROGRAM(VoxGeom)(const void *geomData,
                              box3f &primBounds,
                              const int primID)
{
  const VoxGeomData &self = *(const VoxGeomData*)geomData;
  uchar4 indices = self.prims[primID];
  const vec3f boxmin( indices.x, indices.y, indices.z );
  const vec3f boxmax( 1+indices.x, 1+indices.y, 1+indices.z );
  primBounds = box3f(boxmin, boxmax);
}

inline __device__ float3 makeFaceNormal( float3 t0, float3 t1, float t)
{
    float3 N;
    if (t == t1.x) 
        //+X
        N = make_float3( 1.0f,  0.0f,  0.0f);
    else if (t == t0.x)
        //-X
        N = make_float3( -1.0f,  0.0f,  0.0f);
    else if (t == t1.y)
        //+Y
        N = make_float3( 0.0f,  1.0f,  0.0f);
    else if (t == t0.y)
        //-Y
        N = make_float3( 0.0f, -1.0f,  0.0f);
    else if (t == t1.z)
        //+Z,
        N = make_float3( 0.0f,  0.0f,  1.0f);
    else //if (t == t0.z)
        //-Z
        N = make_float3( 0.0f,  0.0f, -1.0f);

    return N;
}

OPTIX_INTERSECT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  const vec3f boxRadius(0.5f, 0.5f, 0.5f);
  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  vec3f tnear = owl::min(t0, t1);
  vec3f tfar = owl::max(t0, t1);
  float tmin = reduce_max(tnear);
  float tmax = reduce_min(tfar);

  if (tmin <= tmax && tmin > ray_tmin && tmin < ray_tmax) {
    const float3 N = makeFaceNormal(t0, t1, tmin);
    optixReportIntersection( tmin, 0, float_as_int(N.x), float_as_int(N.y), float_as_int(N.z));
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();

  vec3f &prd = owl::getPRD<vec3f>();

  // Select normal for whichever face we hit
  const float3 Nbox = make_float3(
        int_as_float(optixGetAttribute_0()),
        int_as_float(optixGetAttribute_1()),
        int_as_float(optixGetAttribute_2()));
  const vec3f Ng = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Convert 8 bit color to float
  const int ci = self.prims[primID].w;
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  const vec3f rayDir = optixGetWorldRayDirection();
  prd = (.2f + .8f*fabs(dot(rayDir,Ng)))*color;


}

OPTIX_MISS_PROGRAM(miss)()
{
  const vec2i pixelID = owl::getLaunchIndex();

  const MissProgData &self = owl::getProgramData<MissProgData>();
  
  vec3f &prd = owl::getPRD<vec3f>();
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  prd = (pattern&1) ? self.color1 : self.color0;
}

