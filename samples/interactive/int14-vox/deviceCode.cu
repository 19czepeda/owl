#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "constants.h"
#include <optix_device.h>

#include <owl/common/math/random.h>
#include <owl/common/math/LinearSpace.h>


__constant__ LaunchParams optixLaunchParams;


typedef owl::common::LCG<4> Random;
typedef owl::RayT<0, 3> RadianceRay;
typedef owl::RayT<1, 3> ShadowRay;
typedef owl::RayT<2, 3> OutlineShadowRay;

inline __device__
vec3f missColor(const RadianceRay &ray)
{
  const vec2i pixelID = owl::getLaunchIndex();

  const vec3f rayDir = normalize(ray.direction);
  const float t = 0.5f*(rayDir.y + 1.0f);
  const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
  /* nothing to do */
}

OPTIX_MISS_PROGRAM(miss_shadow)()
{
  float &vis = owl::getPRD<float>();
  vis = 1.f;
}

typedef enum {
  /*! ray could get properly bounced, and is still alive */
  rayGotBounced,
  /*! ray didn't hit anything, and went into the environment */
  rayDidntHitAnything
} ScatterEvent;


struct PerRayData
{
  Random random;
  struct {
    ScatterEvent scatterEvent;
    vec3f        scattered_origin;
    vec3f        scattered_direction;
    vec3f        attenuation;
    vec3f        directLight;
    float        hitDistance;
  } out;
};

template <typename RayT, int Mask>
inline __device__
RayT makeRay(const vec3f &origin,
             const vec3f &direction,
             float tmin,
             float tmax)
{
  if (optixLaunchParams.enableClipping) {
    const float eps = 0.01f * optixLaunchParams.brickScale;
    const float clipZ = optixLaunchParams.clipHeight * optixLaunchParams.brickScale;
    const float t = (clipZ - origin.z) / direction.z;
    if (direction.z < 0.f) {
      tmin = owl::max(tmin, t-eps);
    } else {
      tmax = owl::min(tmax, t+eps);
    }
  }

  return RayT(origin, direction, tmin, tmax, Mask);
}



inline __device__
RadianceRay makeRadianceRay(const vec3f &origin, 
                            const vec3f &direction,
                            float tmin,
                            float tmax)
{
  return makeRay<RadianceRay, VISIBILITY_RADIANCE>(origin, direction, tmin, tmax);
}

inline __device__
ShadowRay makeShadowRay(const vec3f &origin, 
                        const vec3f &direction,
                        float tmin,
                        float tmax)
{
  return makeRay<ShadowRay, VISIBILITY_SHADOW>(origin, direction, tmin, tmax);
}

inline __device__
OutlineShadowRay makeOutlineShadowRay(const vec3f &origin, 
                                      const vec3f &direction,
                                      float tmin,
                                      float tmax)
{
  return makeRay<OutlineShadowRay, VISIBILITY_OUTLINE>(origin, direction, tmin, tmax);
}

inline __device__
vec3f tracePrimaryRay(const RayGenData &self,
                RadianceRay &ray, PerRayData &prd, float &firstHitDistance)
{
  if (optixLaunchParams.enableToonOutline)
    firstHitDistance = 1e10f;
  
  prd.out.scatterEvent = rayDidntHitAnything;
  owl::traceRay(/*accel to trace against*/ optixLaunchParams.world,
                /*the ray to trace*/ ray,
                /*prd*/prd,
                OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES);
    
  if (prd.out.scatterEvent == rayDidntHitAnything)
    /* ray got 'lost' to the environment - 'light' it with miss
       shader */
    return missColor(ray);
  else { // ray is still alive, and got properly bounced
    ray = makeRadianceRay(/* origin   : */ prd.out.scattered_origin,
                   /* direction: */ prd.out.scattered_direction,
                   /* tmin     : */ 1e-3f,
                   /* tmax     : */ 1e10f);

    if (optixLaunchParams.enableToonOutline) {
      firstHitDistance = prd.out.hitDistance;
    }
  }
  return prd.out.directLight;
}

inline __device__
vec3f traceBounces(const RayGenData &self,
                RadianceRay &ray, PerRayData &prd)
{
  vec3f attenuation = prd.out.attenuation;
  vec3f directLight = 0.f;

  constexpr int MaxDepth = 2;
  
  /* iterative version of recursion, up to max depth */
  for (int depth=1;depth<MaxDepth;depth++) {
    prd.out.scatterEvent = rayDidntHitAnything;
    owl::traceRay(/*accel to trace against*/ optixLaunchParams.world,
                  /*the ray to trace*/ ray,
                  /*prd*/prd,
                  OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES);
    
    if (prd.out.scatterEvent == rayDidntHitAnything)
      /* ray got 'lost' to the environment - 'light' it with miss
         shader */
      return directLight + attenuation * missColor(ray);
    else { // ray is still alive, and got properly bounced
      attenuation *= prd.out.attenuation;
      directLight += prd.out.directLight;
      ray = makeRadianceRay(/* origin   : */ prd.out.scattered_origin,
                     /* direction: */ prd.out.scattered_direction,
                     /* tmin     : */ 1e-3f,
                     /* tmax     : */ 1e10f);

    }
  }
  // recursion did not terminate - cancel it but return direct lighting from any previous bounce
  return directLight;
}

// returns a visibility term (1 for unshadowed)
inline __device__
float traceShadowRay(const OptixTraversableHandle &traversable,
                  ShadowRay &ray)
{
  float vis = 0.f;
  owl::traceRay(traversable, ray, vis, 
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT
                   | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
                   | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return vis;
}

// returns a visibility term (1 for unshadowed)
inline __device__
float traceOutlineShadowRay(const OptixTraversableHandle &traversable,
                  OutlineShadowRay &ray)
{
  float vis = 0.f;
  owl::traceRay(traversable, ray, vis, 
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT
                   | OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES
                   | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
                   | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return vis;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  const vec2i fbSize = optixLaunchParams.fbSize;
  const int fbIndex = pixelID.x+fbSize.x*pixelID.y;

  PerRayData prd;
  prd.random.init(fbIndex, optixLaunchParams.frameID);

  const int NUM_SAMPLES_PER_PIXEL = 4;
  vec3f accumColor = 0.f;

  for (int sampleID=0;sampleID<NUM_SAMPLES_PER_PIXEL;sampleID++) {

    const vec2f pixelSample(prd.random(), prd.random());
    const vec2f screen = (vec2f(pixelID)+pixelSample) / vec2f(fbSize);

    const vec3f rayDir = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);

    RadianceRay ray = makeRadianceRay(self.camera.pos, rayDir, 0.f, 1e30f);

    float firstHitDistance = 1e10f;
    vec3f color = tracePrimaryRay(self, ray, prd, firstHitDistance);

    float visibility = 1.f;
    if (optixLaunchParams.enableToonOutline) {

      // Feature size control for outlines
      const float outlineDepthBias = 5*optixLaunchParams.brickScale;
      OutlineShadowRay outlineShadowRay = makeOutlineShadowRay(self.camera.pos,
          rayDir, 0.f, firstHitDistance-outlineDepthBias);
      visibility = traceOutlineShadowRay(optixLaunchParams.world, outlineShadowRay);
    }
    if (visibility > 0.f) {
      if (prd.out.scatterEvent == rayGotBounced) {
        color += traceBounces(self, ray, prd);
      } 
      accumColor += color*visibility;
    }
  }
    
  vec4f rgba {accumColor / NUM_SAMPLES_PER_PIXEL, 1.0f};

  if (optixLaunchParams.frameID > 0) {
    // Blend with accum buffer
    const vec4f accum = optixLaunchParams.fbAccumBuffer[fbIndex];
    rgba += float(optixLaunchParams.frameID) * accum; 
    rgba /= (optixLaunchParams.frameID+1.f);
  }

  optixLaunchParams.fbAccumBuffer[fbIndex] = (float4)rgba;
  optixLaunchParams.fbPtr[fbIndex] = owl::make_rgba(rgba);
}

inline __device__ 
float2 squareToDisk(float u1, float u2)
{
  // Uniformly sample disk.
  const float r   = sqrtf( u1 );
  const float phi = 2.0f*M_PIf * u2;
  float2 p = {r * cosf( phi ), r * sinf( phi )};
  return p;
}

inline __device__
vec3f cosineSampleHemisphere(float u1, float u2)
{
  float2 p = squareToDisk(u1, u2);

  // Project up to hemisphere.
  return vec3f(p.x, p.y, sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) ));
}

inline __device__
vec3f sunlight(const vec3f &hit_P_offset, const vec3f &Ng, Random &random)
{
  const vec3f lightDir = optixLaunchParams.sunDirection;

  const float NdotL = dot(Ng, lightDir);
  if (NdotL <= 0.f) {
    return 0.f;  // below horizon
  }

  // Build frame around light dir
  const owl::LinearSpace3f lightFrame = owl::common::frame(normalize(lightDir));

  // jitter light direction slightly
  const float lightRadius = 0.01f;  // should be ok as a constant since our scenes are normalized
  const vec3f lightCenter = hit_P_offset + lightFrame.vz;
  const float2 sample = squareToDisk(random(), random());
  const vec3f jitteredPos = lightCenter + lightRadius*(sample.x*lightFrame.vx + sample.y*lightFrame.vy);
  const vec3f jitteredLightDir = jitteredPos - hit_P_offset;  // no need to normalize

  ShadowRay shadowRay = makeShadowRay(hit_P_offset, jitteredLightDir, 0.f, 1e10f);
  float vis = traceShadowRay(optixLaunchParams.world, shadowRay);
  return vis * optixLaunchParams.sunColor * NdotL; 

}

inline __device__ 
vec3f scatterLambertian(const vec3f &Ng, Random &random)
{
  const owl::LinearSpace3f shadingFrame = owl::common::frame(Ng);
  vec3f scatteredDirectionInShadingFrame = cosineSampleHemisphere(random(), random());
  vec3f scatteredDirection = shadingFrame.vx * scatteredDirectionInShadingFrame.x +
                             shadingFrame.vy * scatteredDirectionInShadingFrame.y +
                             shadingFrame.vz * scatteredDirectionInShadingFrame.z;

  return scatteredDirection;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Nbox   = normalize(cross(B-A,C-A));
  const vec3f Ng     = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Bias value relative to a brick
  const float shadowBias = 1e-2f * fminf(1.f, optixLaunchParams.brickScale);

  // Convert 8 bit color to float
  const unsigned int brickID = self.isFlat ? optixGetPrimitiveIndex() / self.primCountPerBrick : optixGetInstanceId();
  const int ci = self.colorIndexPerBrick[brickID];
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  PerRayData &prd = owl::getPRD<PerRayData>();
  const vec3f dir   = optixGetWorldRayDirection();
  const vec3f org   = optixGetWorldRayOrigin();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f hit_P_offset = hit_P + shadowBias*Ng;  // bias along normal to help with shadow acne

  // Direct
  const vec3f directLight = sunlight(hit_P_offset, Ng, prd.random);

  // Bounce
  vec3f scatteredDirection = scatterLambertian(Ng, prd.random);

  prd.out.directLight = directLight*color;
  prd.out.attenuation = color;
  prd.out.scatterEvent = rayGotBounced;
  prd.out.scattered_direction = scatteredDirection;
  prd.out.scattered_origin = hit_P_offset;

  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = length(hit_P - org);
  }

}

OPTIX_BOUNDS_PROGRAM(VoxGeom)(const void *geomData,
                              box3f &primBounds,
                              const int primID)
{
  const VoxGeomData &self = *(const VoxGeomData*)geomData;
  uchar4 indices = self.prims[primID];
  vec3f boxmin( indices.x, indices.y, indices.z );
  vec3f boxmax( 1+indices.x, 1+indices.y, 1+indices.z );

  if (self.enableToonOutline) {
    // bloat the box slightly
    const vec3f boxcenter (indices.x + 0.5f, indices.y + 0.5f, indices.z + 0.5f);
    boxmin = boxcenter + OUTLINE_SCALE*(boxmin-boxcenter);
    boxmax = boxcenter + OUTLINE_SCALE*(boxmax-boxcenter);
  }
  
  primBounds = box3f(boxmin, boxmax);
}

OPTIX_INTERSECT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  const vec3f boxRadius(0.5f, 0.5f, 0.5f);
  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  float tnear = reduce_max(owl::min(t0, t1));
  float tfar  = reduce_min(owl::max(t0, t1));

  // Only handle the case where the ray starts outside the box

  if (tnear <= tfar && tnear > ray_tmin && tnear < ray_tmax) {
    // compute face normal at local hit point
    vec3f V = rayOrigin + rayDirection*tnear;
    vec3f N(0.0f);
    int i = indexOfMaxComponent(abs(V));
    N[i] = (V[i] >= 0.0f) ? 1 : -1;
    optixReportIntersection( tnear, 0, float_as_int(N.x), float_as_int(N.y), float_as_int(N.z));
  }
}

// Used for toon outline 
OPTIX_INTERSECT_PROGRAM(VoxGeomShadowCullFront)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  if (!self.enableToonOutline) {
      return;
  }
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  const float outlinePad = 1.2f;  // needs to match bounding box program
  const vec3f boxRadius = vec3f(0.5f, 0.5f, 0.5f)*outlinePad;
  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  float tnear = reduce_max(owl::min(t0, t1));
  float tfar  = reduce_min(owl::max(t0, t1));

  // Cull front face by using tfar for the hit

  if (tnear <= tfar && tfar > ray_tmin && tfar < ray_tmax) {
    optixReportIntersection( tfar, 0);
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();

  // Select normal for whichever face we hit
  const float3 Nbox = make_float3(
        int_as_float(optixGetAttribute_0()),
        int_as_float(optixGetAttribute_1()),
        int_as_float(optixGetAttribute_2()));
  const vec3f Ng = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Bias value relative to brick scale
  const float shadowBias = 1e-2f * fminf(1.f, optixLaunchParams.brickScale);

  // Convert 8 bit color to float
  const int ci = self.prims[primID].w;
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  PerRayData &prd = owl::getPRD<PerRayData>();
  const vec3f dir   = optixGetWorldRayDirection();
  const vec3f org   = optixGetWorldRayOrigin();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f hit_P_offset = hit_P + shadowBias*Ng;  // bias along normal to help with shadow acne

  // Direct
  const vec3f directLight = sunlight(hit_P_offset, Ng, prd.random);

  // Bounce
  vec3f scatteredDirection = scatterLambertian(Ng, prd.random);

  prd.out.directLight = directLight*color;
  prd.out.attenuation = color;
  prd.out.scatterEvent = rayGotBounced;
  prd.out.scattered_direction = scatteredDirection;
  prd.out.scattered_origin = hit_P_offset;

  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = length(hit_P - org);
  }

}


