#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019-2020 Ingo Wald                                            //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include "constants.h"
#include <optix_device.h>

#include <owl/common/math/random.h>
#include <owl/common/math/LinearSpace.h>


__constant__ LaunchParams optixLaunchParams;


typedef owl::common::LCG<4> Random;
typedef owl::RayT<0, 3> RadianceRay;
typedef owl::RayT<1, 3> ShadowRay;
typedef owl::RayT<2, 3> OutlineShadowRay;

inline __device__
vec3f missColor(const RadianceRay &ray)
{
  const vec2i pixelID = owl::getLaunchIndex();

  const vec3f rayDir = normalize(ray.direction);
  const float t = 0.5f*(rayDir.y + 1.0f);
  const vec3f c = (1.0f - t)*vec3f(1.0f, 1.0f, 1.0f) + t * vec3f(0.5f, 0.7f, 1.0f);
  return c;
}

OPTIX_MISS_PROGRAM(miss)()
{
  /* nothing to do */
}

OPTIX_MISS_PROGRAM(miss_shadow)()
{
  float &vis = owl::getPRD<float>();
  vis = 1.f;
}

typedef enum {
  /*! ray could get properly bounced, and is still alive */
  rayGotBounced,
  /*! ray didn't hit anything, and went into the environment */
  rayDidntHitAnything
} ScatterEvent;


struct PerRayData
{
  Random random;
  struct {
    ScatterEvent scatterEvent;
    vec3f        scattered_origin;
    vec3f        scattered_direction;
    vec3f        attenuation;
    vec3f        directLight;
    float        hitDistance;
  } out;
};

template <typename RayT, int Mask>
inline __device__
RayT makeRay(const vec3f &origin,
             const vec3f &direction,
             float tmin,
             float tmax)
{
  if (optixLaunchParams.enableClipping) {
    const float eps = 0.01f * optixLaunchParams.brickScale;
    const float clipZ = optixLaunchParams.clipHeight * optixLaunchParams.brickScale;
    const float t = (clipZ - origin.z) / direction.z;
    if (direction.z < 0.f) {
      tmin = owl::max(tmin, t-eps);
    } else {
      tmax = owl::min(tmax, t+eps);
    }
  }

  return RayT(origin, direction, tmin, tmax, Mask);
}



inline __device__
RadianceRay makeRadianceRay(const vec3f &origin, 
                            const vec3f &direction,
                            float tmin,
                            float tmax)
{
  return makeRay<RadianceRay, VISIBILITY_RADIANCE>(origin, direction, tmin, tmax);
}

inline __device__
ShadowRay makeShadowRay(const vec3f &origin, 
                        const vec3f &direction,
                        float tmin,
                        float tmax)
{
  return makeRay<ShadowRay, VISIBILITY_SHADOW>(origin, direction, tmin, tmax);
}

inline __device__
OutlineShadowRay makeOutlineShadowRay(const vec3f &origin, 
                                      const vec3f &direction,
                                      float tmin,
                                      float tmax)
{
  return makeRay<OutlineShadowRay, VISIBILITY_OUTLINE>(origin, direction, tmin, tmax);
}

inline __device__
vec3f tracePrimaryRay(const RayGenData &self,
                RadianceRay &ray, PerRayData &prd)
{
  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = 1e10f;
  }
  
  prd.out.scatterEvent = rayDidntHitAnything;
  owl::traceRay(optixLaunchParams.world,
                ray,
                prd,
                OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES);
    
  if (prd.out.scatterEvent == rayDidntHitAnything) {
    return missColor(ray);
  }
  else { // ray is still alive, and got properly bounced
    ray = makeRadianceRay(prd.out.scattered_origin,
                          prd.out.scattered_direction,
                          0.f, // rely on hitP offset along normal
                          1e10f);
  }
  return prd.out.directLight;
}

inline __device__
vec3f traceBounces(const RayGenData &self,
                RadianceRay &ray, PerRayData &prd)
{
  vec3f attenuation = prd.out.attenuation;
  vec3f directLight = 0.f;

  constexpr int MaxDepth = 2;
  
  /* iterative version of recursion, up to max depth */
  for (int depth=1;depth<MaxDepth;depth++) {
    prd.out.scatterEvent = rayDidntHitAnything;
    owl::traceRay(/*accel to trace against*/ optixLaunchParams.world,
                  /*the ray to trace*/ ray,
                  /*prd*/prd,
                  OPTIX_RAY_FLAG_CULL_BACK_FACING_TRIANGLES);
    
    if (prd.out.scatterEvent == rayDidntHitAnything)
      /* ray got 'lost' to the environment - 'light' it with miss
         shader */
      return directLight + attenuation * missColor(ray);
    else { // ray is still alive, and got properly bounced
      attenuation *= prd.out.attenuation;
      directLight += prd.out.directLight;
      ray = makeRadianceRay(/* origin   : */ prd.out.scattered_origin,
                     /* direction: */ prd.out.scattered_direction,
                     /* tmin     : */ 0.f,  // rely on hitP offset along normal
                     /* tmax     : */ 1e10f);

    }
  }
  // recursion did not terminate - cancel it but return direct lighting from any previous bounce
  return directLight;
}

// returns a visibility term (1 for unshadowed)
inline __device__
float traceShadowRay(const OptixTraversableHandle &traversable,
                  ShadowRay &ray)
{
  float vis = 0.f;
  owl::traceRay(traversable, ray, vis, 
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT
                   | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
                   | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return vis;
}

// returns a visibility term (1 for unshadowed)
inline __device__
float traceOutlineShadowRay(const OptixTraversableHandle &traversable,
                  OutlineShadowRay &ray)
{
  float vis = 0.f;
  owl::traceRay(traversable, ray, vis, 
                   OPTIX_RAY_FLAG_DISABLE_ANYHIT
                   | OPTIX_RAY_FLAG_CULL_FRONT_FACING_TRIANGLES
                   | OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT
                   | OPTIX_RAY_FLAG_DISABLE_CLOSESTHIT);
  return vis;
}

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  const RayGenData &self = owl::getProgramData<RayGenData>();
  const vec2i pixelID = owl::getLaunchIndex();
  const vec2i fbSize = optixLaunchParams.fbSize;
  const int fbIndex = pixelID.x+fbSize.x*pixelID.y;

  PerRayData prd;
  prd.random.init(fbIndex, optixLaunchParams.frameID);

  // Note: measured to be faster to keep the loop over subpixels at the launch level,
  // not inside raygen.  Maybe a long tail effect?
  constexpr int NUM_SAMPLES_PER_PIXEL = 1;
  vec3f accumColor = 0.f;

  for (int sampleID=0;sampleID<NUM_SAMPLES_PER_PIXEL;sampleID++) {

    const vec2f pixelSample(prd.random(), prd.random());
    const vec2f screen = (vec2f(pixelID)+pixelSample) / vec2f(fbSize);

    const vec3f rayDir = normalize(self.camera.dir_00
                  + screen.u * self.camera.dir_du
                  + screen.v * self.camera.dir_dv);

    RadianceRay ray = makeRadianceRay(self.camera.pos, rayDir, 0.f, 1e30f);

    vec3f color = tracePrimaryRay(self, ray, prd);

    float visibility = 1.f;
    if (optixLaunchParams.enableToonOutline) {

      // Feature size control for outlines
      const float outlineDepthBias = 5*optixLaunchParams.brickScale;
      const float firstHitDistance = prd.out.hitDistance; // Note: dependency on primary ray
      OutlineShadowRay outlineShadowRay = makeOutlineShadowRay(self.camera.pos,
          rayDir, 0.f, firstHitDistance-outlineDepthBias);
      visibility = traceOutlineShadowRay(optixLaunchParams.world, outlineShadowRay);
    }
    // Note: measurable speedup from tracing bounce rays unconditionally without
    // checking visibility first.
    if (prd.out.scatterEvent == rayGotBounced) {
      color += traceBounces(self, ray, prd);
    } 
    accumColor += color*visibility;
  }
    
  vec4f rgba {accumColor / NUM_SAMPLES_PER_PIXEL, 1.0f};

  if (optixLaunchParams.frameID > 0) {
    // Blend with accum buffer
    const vec4f accum = optixLaunchParams.fbAccumBuffer[fbIndex];
    rgba += float(optixLaunchParams.frameID) * accum; 
    rgba /= (optixLaunchParams.frameID+1.f);
  }

  optixLaunchParams.fbAccumBuffer[fbIndex] = (float4)rgba;
  optixLaunchParams.fbPtr[fbIndex] = owl::make_rgba(rgba);
}

inline __device__ 
float2 squareToDisk(float u1, float u2)
{
  // Uniformly sample disk.
  const float r   = sqrtf( u1 );
  const float phi = 2.0f*M_PIf * u2;
  float2 p = {r * cosf( phi ), r * sinf( phi )};
  return p;
}

inline __device__
vec3f cosineSampleHemisphere(float u1, float u2)
{
  float2 p = squareToDisk(u1, u2);

  // Project up to hemisphere.
  return vec3f(p.x, p.y, sqrtf( fmaxf( 0.0f, 1.0f - p.x*p.x - p.y*p.y ) ));
}

inline __device__
vec3f sunlight(const vec3f &hit_P_offset, const vec3f &Ng, Random &random)
{
  const vec3f lightDir = optixLaunchParams.sunDirection;

  const float NdotL = dot(Ng, lightDir);
  if (NdotL <= 0.f) {
    return 0.f;  // below horizon
  }

  // Build frame around light dir
  const owl::LinearSpace3f lightFrame = owl::common::frame(normalize(lightDir));

  // jitter light direction slightly
  const float lightRadius = 0.01f;  // should be ok as a constant since our scenes are normalized
  const vec3f lightCenter = hit_P_offset + lightFrame.vz;
  const float2 sample = squareToDisk(random(), random());
  const vec3f jitteredPos = lightCenter + lightRadius*(sample.x*lightFrame.vx + sample.y*lightFrame.vy);
  const vec3f jitteredLightDir = jitteredPos - hit_P_offset;  // no need to normalize

  ShadowRay shadowRay = makeShadowRay(hit_P_offset, jitteredLightDir, 0.f, 1e10f);
  float vis = traceShadowRay(optixLaunchParams.world, shadowRay);
  return vis * optixLaunchParams.sunColor * NdotL; 

}

inline __device__ 
vec3f scatterLambertian(const vec3f &Ng, Random &random)
{
  const owl::LinearSpace3f shadingFrame = owl::common::frame(Ng);
  vec3f scatteredDirectionInShadingFrame = cosineSampleHemisphere(random(), random());
  vec3f scatteredDirection = shadingFrame.vx * scatteredDirectionInShadingFrame.x +
                             shadingFrame.vy * scatteredDirectionInShadingFrame.y +
                             shadingFrame.vz * scatteredDirectionInShadingFrame.z;

  return scatteredDirection;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  const TrianglesGeomData &self = owl::getProgramData<TrianglesGeomData>();
  
  // compute normal:
  const int   primID = optixGetPrimitiveIndex();
  const vec3i index  = self.index[primID];
  const vec3f &A     = self.vertex[index.x];
  const vec3f &B     = self.vertex[index.y];
  const vec3f &C     = self.vertex[index.z];
  const vec3f Nbox   = normalize(cross(B-A,C-A));
  const vec3f Ng     = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Bias value relative to a brick
  const float shadowBias = 0.01f * fminf(1.f, optixLaunchParams.brickScale);

  // Convert 8 bit color to float
  const unsigned int brickID = self.isFlat ? optixGetPrimitiveIndex() / self.primCountPerBrick : optixGetInstanceId();
  const int ci = self.colorIndexPerBrick[brickID];
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  PerRayData &prd = owl::getPRD<PerRayData>();
  const vec3f dir   = optixGetWorldRayDirection();
  const vec3f org   = optixGetWorldRayOrigin();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f hit_P_offset = hit_P + shadowBias*Ng;  // bias along normal to help with shadow acne

  // Direct
  const vec3f directLight = sunlight(hit_P_offset, Ng, prd.random);

  // Bounce
  vec3f scatteredDirection = scatterLambertian(Ng, prd.random);

  prd.out.directLight = directLight*color;
  prd.out.attenuation = color;
  prd.out.scatterEvent = rayGotBounced;
  prd.out.scattered_direction = scatteredDirection;
  prd.out.scattered_origin = hit_P_offset;

  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = length(hit_P - org);
  }

}

OPTIX_BOUNDS_PROGRAM(VoxGeom)(const void *geomData,
                              box3f &primBounds,
                              const int primID)
{
  const VoxGeomData &self = *(const VoxGeomData*)geomData;
  uchar4 indices = self.prims[primID];
  vec3f boxmin( indices.x, indices.y, indices.z );
  vec3f boxmax( 1+indices.x, 1+indices.y, 1+indices.z );

  if (self.enableToonOutline) {
    // bloat the box slightly
    const vec3f boxcenter (indices.x + 0.5f, indices.y + 0.5f, indices.z + 0.5f);
    boxmin = boxcenter + OUTLINE_SCALE*(boxmin-boxcenter);
    boxmax = boxcenter + OUTLINE_SCALE*(boxmax-boxcenter);
  }
  
  primBounds = box3f(boxmin, boxmax);
}



namespace {
  // temp swizzles
  inline __device__
  vec2f yz(vec3f v)
  {
    return vec2f(v.y, v.z);
  }
  inline __device__
  vec2f zx(vec3f v)
  {
    return vec2f(v.z, v.x);
  }
  inline __device__
  vec2f xy(vec3f v)
  {
    return vec2f(v.x, v.y);
  }
}

// Ray-box intersection with normals from Majercik et al 2018
OPTIX_INTERSECT_PROGRAM(VoxGeomMajercik)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  // Negated sign function
  const vec3f sgn( 
      rayDirection.x > 0.f ? -1 : 1,
      rayDirection.y > 0.f ? -1 : 1,
      rayDirection.z > 0.f ? -1 : 1);

  const vec3f boxRadius(0.5f, 0.5f, 0.5f);
  vec3f distanceToPlane = boxRadius*sgn - rayOrigin;
  distanceToPlane *= invRayDirection;

  const bool testX = distanceToPlane.x >= 0.f && 
    owl::all_less_than(owl::abs(yz(rayOrigin) + yz(rayDirection)*distanceToPlane.x), yz(boxRadius));

  const bool testY = distanceToPlane.y >= 0.f &&
    owl::all_less_than(owl::abs(zx(rayOrigin) + zx(rayDirection)*distanceToPlane.y), zx(boxRadius));

  const bool testZ = distanceToPlane.z >= 0.f &&
    owl::all_less_than(owl::abs(xy(rayOrigin) + xy(rayDirection)*distanceToPlane.z), xy(boxRadius));

  const vec3b test(testX, testY, testZ);
  if ( test.x || test.y || test.z ) { // hit the box
    float distance = test.x ? distanceToPlane.x : (test.y ? distanceToPlane.y : distanceToPlane.z);
    const float ray_tmax = optixGetRayTmax();
    const float ray_tmin = optixGetRayTmin();
    if (distance > ray_tmin && distance < ray_tmax) {  // closer than existing hit
      // Since N is something like [0,-1,0], encode it as sign (1 bit) and 3 components (3 bits): 000...SNNN
      // This lets it fit in one attribute.
      int signOfN = (sgn.x*test.x + sgn.y*test.y + sgn.z*test.z) > 0 ? 1 : 0;
      int packedN = (signOfN << 3) | (test.z << 2) | (test.y << 1) | test.x;
      optixReportIntersection(distance, 0, packedN);
    }
  }
}

// "Efficient slabs" method
OPTIX_INTERSECT_PROGRAM(VoxGeomShadow)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  const vec3f boxRadius(0.5f, 0.5f, 0.5f);
  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  float tnear = reduce_max(owl::min(t0, t1));
  float tfar  = reduce_min(owl::max(t0, t1));
  float distance = tnear > 0.f ? tnear : tfar;

  if (tnear <= tfar && distance > ray_tmin && distance < ray_tmax) {
    optixReportIntersection( distance, 0);
  }
}

// Used for toon outline where we don't need normals, and cull front faces.
OPTIX_INTERSECT_PROGRAM(VoxGeomOutlineShadow)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();
  if (!self.enableToonOutline) {
      return;
  }
  uchar4 indices = self.prims[primID];
  vec3f boxCenter(indices.x+0.5, indices.y+0.5, indices.z+0.5);

  // Translate ray to local box space
  const vec3f rayOrigin  = vec3f(optixGetObjectRayOrigin()) - boxCenter;
  const vec3f rayDirection  = optixGetObjectRayDirection();  // assume no rotation
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const float ray_tmax = optixGetRayTmax();
  const float ray_tmin = optixGetRayTmin();

  const vec3f boxRadius = vec3f(0.5f, 0.5f, 0.5f)*OUTLINE_SCALE;
  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  float tnear = reduce_max(owl::min(t0, t1));
  float tfar  = reduce_min(owl::max(t0, t1));

  // Cull front face by using tfar for the hit

  if (tnear <= tfar && tfar > ray_tmin && tfar < ray_tmax) {
    optixReportIntersection( tfar, 0);
  }
}

OPTIX_CLOSEST_HIT_PROGRAM(VoxGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxGeomData &self = owl::getProgramData<VoxGeomData>();

  // Select normal for whichever face we hit
  const int packedN = optixGetAttribute_0();
  const int sgnN = (packedN >> 3) ? 1 : -1;
  const float3 Nbox = make_float3(
    sgnN * ( packedN       & 1),
    sgnN * ((packedN >> 1) & 1),
    sgnN * ((packedN >> 2) & 1));

  const vec3f Ng = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Bias value relative to brick scale
  const float shadowBias = 1e-2f * fminf(1.f, optixLaunchParams.brickScale);

  // Convert 8 bit color to float
  const int ci = self.prims[primID].w;
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  PerRayData &prd = owl::getPRD<PerRayData>();
  const vec3f dir   = optixGetWorldRayDirection();
  const vec3f org   = optixGetWorldRayOrigin();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f hit_P_offset = hit_P + shadowBias*Ng;  // bias along normal to help with shadow acne

  // Direct
  const vec3f directLight = sunlight(hit_P_offset, Ng, prd.random);

  // Bounce
  vec3f scatteredDirection = scatterLambertian(Ng, prd.random);

  prd.out.directLight = directLight*color;
  prd.out.attenuation = color;
  prd.out.scatterEvent = rayGotBounced;
  prd.out.scattered_direction = scatteredDirection;
  prd.out.scattered_origin = hit_P_offset;

  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = length(hit_P - org);
  }

}


// Experiment with VoxBlockGeom


OPTIX_BOUNDS_PROGRAM(VoxBlockGeom)(const void *geomData,
                              box3f &primBounds,
                              const int primID)
{
  const VoxBlockGeomData &self = *(const VoxBlockGeomData*)geomData;
  uchar3 indices = self.prims[primID];
  vec3f boxmin( indices.x, indices.y, indices.z );
  vec3f boxmax = boxmin + vec3f(BLOCKLEN);

  // Not obvious how to toon outline
  /*
  if (self.enableToonOutline) {
    // bloat the box slightly
    const vec3f boxcenter (indices.x + 0.5f, indices.y + 0.5f, indices.z + 0.5f);
    boxmin = boxcenter + OUTLINE_SCALE*(boxmin-boxcenter);
    boxmax = boxcenter + OUTLINE_SCALE*(boxmax-boxcenter);
  }
  */
  
  primBounds = box3f(boxmin, boxmax);
}

// Note: ignore ray span here.
__device__ inline
bool intersectRayBox(const vec3f _rayOrigin, const vec3f rayDirection,
    vec3f boxCenter, vec3f boxRadius,
    float &tnear, float &tfar)
{
  // Translate ray to local box space
  const vec3f rayOrigin  = _rayOrigin - boxCenter;
  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  vec3f t0 = (-boxRadius - rayOrigin) * invRayDirection;
  vec3f t1 = ( boxRadius - rayOrigin) * invRayDirection;
  tnear = reduce_max(owl::min(t0, t1));
  tfar  = reduce_min(owl::max(t0, t1));
  return tnear <= tfar;
}

template <bool IsShadowRay>
inline __device__ 
void intersectVoxBlockGeom()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxBlockGeomData &self = owl::getProgramData<VoxBlockGeomData>();

  const uchar3 blockOrigin = self.prims[primID];
  const vec3f blockRadius(0.5f*BLOCKLEN);
  const vec3f blockCenter = vec3f(blockOrigin.x, blockOrigin.y, blockOrigin.z) + blockRadius;

  const vec3f rayOrigin = optixGetObjectRayOrigin();
  const vec3f rayDirection = optixGetObjectRayDirection();
  const float rayTmin = optixGetRayTmin();
  const float rayTmax = optixGetRayTmax();

  const vec3f blockOrigin3f(blockOrigin.x, blockOrigin.y, blockOrigin.z);

  int axis = -1;  // axis we crossed most recently, which gives the normal
  vec3f cell3f;
  float tnear;    // init where ray enters block, and increases as we traverse cells
  float tfar;
  if (!intersectRayBox(rayOrigin, rayDirection, blockCenter, blockRadius, tnear, tfar)) {
    return;  // Ray line misses block (without considering ray span)
  }

  // Apply ray span
  if (tnear >= rayTmin && tnear <= rayTmax) {
    // Ray starts outside and hits block
    cell3f = rayOrigin + tnear*rayDirection - blockOrigin3f;
    axis = indexOfMaxComponent(owl::abs(cell3f - blockRadius)); 

  } else if (tnear < rayTmin && tfar > rayTmin) {
    // Ray starts inside block
    tnear = rayTmin;
    cell3f = (rayOrigin + tnear*rayDirection) - blockOrigin3f;
  } else {
    // Ray does not intersect block within [min,max] span
    return;
  }

  // DDA from PBRT/scratchapixel
  
  // Constants during traversal

  const vec3i blockDim(BLOCKLEN);

  const vec3f sgn( 
      rayDirection.x > 0.f ? 1 : -1,
      rayDirection.y > 0.f ? 1 : -1,
      rayDirection.z > 0.f ? 1 : -1);

  const vec3f invRayDirection = vec3f(1.0f) / rayDirection;

  const vec3f deltaT = sgn * invRayDirection;
  const vec3i step (sgn.x, sgn.y, sgn.z);
  const vec3i exitCell ( 
      sgn.x < 0 ? -1 : blockDim.x,
      sgn.y < 0 ? -1 : blockDim.y,
      sgn.z < 0 ? -1 : blockDim.z);

  const int map[8] = {2, 1, 2, 1, 2, 2, 0, 0};   // TODO: const mem?
  const int brickOffset = primID*blockDim.x*blockDim.y*blockDim.z;


  // Things that change during traversal

  vec3i cell (
      clamp(int(cell3f.x), 0, blockDim.x-1),
      clamp(int(cell3f.y), 0, blockDim.y-1),
      clamp(int(cell3f.z), 0, blockDim.z-1));

  vec3f nextCrossingT (
      tnear + ((sgn.x < 0 ? cell.x : cell.x+1) - cell3f.x) * invRayDirection.x,
      tnear + ((sgn.y < 0 ? cell.y : cell.y+1) - cell3f.y) * invRayDirection.y,
      tnear + ((sgn.z < 0 ? cell.z : cell.z+1) - cell3f.z) * invRayDirection.z);

  int packedNormalForClosestHit = 0; // for radiance rays
  int colorIndexForClosestHit = 0;

  // DDA traversal
  while(1) {
    const int brickIdx = brickOffset + cell.x + cell.y*blockDim.x + cell.z*blockDim.x*blockDim.y;
    const int colorIdx = self.colorIndices[brickIdx];

    // Note: we might have a valid color here but not a valid axis (normal).  This happens when the origin of a bounce
    // ray is pushed inside a neighbor brick due to biasing along the normal.  It is difficult to completely eliminate;
    // even for a flat plane of bricks, the normals on the edges of bricks may point along the plane tangents
    // (think of tiny bevels on the bricks).
    //
    // Extra check for axis >= 0 is a workaround for this.
    
    if (colorIdx > 0 && axis >= 0) {
      if (!IsShadowRay) {
        int signOfNormal = (rayDirection[axis] > 0.f) ? 0 : 1; // here 0 means negative, 1 means positive
        int packedNormal = (signOfNormal << 3) | (1 << axis);
        packedNormalForClosestHit = packedNormal;
      }
      colorIndexForClosestHit = colorIdx;
      break;
    }

    // Advance to next cell along ray

    // Lookup table method from PBRT/scratchapixel, not measured for perf
    const uint8_t k = ((nextCrossingT[0] < nextCrossingT[1]) << 2) + 
                      ((nextCrossingT[0] < nextCrossingT[2]) << 1) + 
                      ((nextCrossingT[1] < nextCrossingT[2])); 

    axis = map[k];
    if (nextCrossingT[axis] >= rayTmax) break;
    cell[axis] += step[axis];
    if (cell[axis] == exitCell[axis]) break;
    tnear = nextCrossingT[axis];
    nextCrossingT[axis] += deltaT[axis];

  }

  if (colorIndexForClosestHit) {
    if (IsShadowRay) {
      optixReportIntersection(tnear, 0);
    } else {
      optixReportIntersection(tnear, 0, packedNormalForClosestHit, colorIndexForClosestHit);
    }
  }

}

OPTIX_INTERSECT_PROGRAM(VoxBlockGeom)()
{
  intersectVoxBlockGeom</*IsShadow=*/ false>();
}

OPTIX_INTERSECT_PROGRAM(VoxBlockGeomShadow)()
{
  intersectVoxBlockGeom</*IsShadow=*/ true>();
}

inline __device__ 
vec3f unpackNormal(int packedN)
{
  const int sgnN = (packedN >> 3) ? 1 : -1;
  const float3 Nbox = make_float3(
    sgnN * ( packedN       & 1),
    sgnN * ((packedN >> 1) & 1),
    sgnN * ((packedN >> 2) & 1));
  return Nbox;
}

OPTIX_CLOSEST_HIT_PROGRAM(VoxBlockGeom)()
{
  // convert indices to 3d box
  const int primID = optixGetPrimitiveIndex();
  const VoxBlockGeomData &self = owl::getProgramData<VoxBlockGeomData>();

  // Select normal for whichever face we hit
  const int packedN = optixGetAttribute_0();
  const vec3f Nbox = unpackNormal(packedN);
  const vec3f Ng = normalize(vec3f(optixTransformNormalFromObjectToWorldSpace(Nbox)));

  // Bias value relative to brick scale
  const float shadowBias = 1e-2f * fminf(1.f, optixLaunchParams.brickScale);

  // Convert 8 bit color to float
  const int ci = optixGetAttribute_1();
  uchar4 col = self.colorPalette[ci];
  const vec3f color = vec3f(col.x, col.y, col.z) * (1.0f/255.0f);

  PerRayData &prd = owl::getPRD<PerRayData>();
  const vec3f dir   = optixGetWorldRayDirection();
  const vec3f org   = optixGetWorldRayOrigin();
  const float hit_t = optixGetRayTmax();
  const vec3f hit_P = org + hit_t * dir;
  const vec3f hit_P_offset = hit_P + shadowBias*Ng;  // bias along normal to help with shadow acne

  // Direct
  const vec3f directLight = sunlight(hit_P_offset, Ng, prd.random);

  // Bounce
  vec3f scatteredDirection = scatterLambertian(Ng, prd.random);

  prd.out.directLight = directLight*color;
  prd.out.attenuation = color;
  prd.out.scatterEvent = rayGotBounced;
  prd.out.scattered_direction = scatteredDirection;
  prd.out.scattered_origin = hit_P_offset;

  // Not used for BlockGeom
  if (optixLaunchParams.enableToonOutline) {
    prd.out.hitDistance = length(hit_P - org);
  }
}

