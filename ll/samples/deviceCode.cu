#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"
#include <optix_device.h>

#define OPTIX_RAYGEN_PROGRAM(programName) \
  extern "C" __global__ \
  void __raygen__##programName

#define OPTIX_CLOSEST_HIT_PROGRAM(programName) \
  extern "C" __global__ \
  void __closesthit__##programName

#define OPTIX_MISS_PROGRAM(programName) \
  extern "C" __global__ \
  void __miss__##programName

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  // RayGenData &rgData = *(RayGenData*)optix::getProgramDataPointer();
  const RayGenData &rg = optix::getProgramData<RayGenData>();
  const vec2i pixelID = optix::getLaunchIndex();
  if (pixelID == optix::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program (on device %i/%i)%s\n",
           GDT_TERMINAL_LIGHT_RED,
           rg.deviceIndex,
           rg.deviceCount,
           GDT_TERMINAL_DEFAULT);
  }
  if (pixelID.x >= rg.fbSize.x) return;
  if (pixelID.y >= rg.fbSize.y) return;

// static __forceinline__ __device__ void optixTrace( OptixTraversableHandle handle,
//                                                    float3                 rayOrigin,
//                                                    float3                 rayDirection,
//                                                    float                  tmin,
//                                                    float                  tmax,
//                                                    float                  rayTime,
//                                                    OptixVisibilityMask    visibilityMask,
//                                                    unsigned int           rayFlags,
//                                                    unsigned int           SBToffset,
//                                                    unsigned int           SBTstride,
//                                                    unsigned int           missSBTIndex,
//                                                    unsigned int&          p0,
//                                                    unsigned int&          p1 );

  const int numRayTypes = 1;
  const int rayType = 0;

  OptixTraversableHandle handle = rg.world;
  float3                 rayOrigin = make_float3(0,0,0);
    float3                 rayDirection = make_float3(0,1,0);
    float                  tmin = 1e-3f;
    float                  tmax = 1e+10f;
    float                  rayTime = 0.f;
    OptixVisibilityMask    visibilityMask = 0;
    unsigned int           rayFlags = 0;
    unsigned int           SBToffset = rayType;
    unsigned int           SBTstride = numRayTypes;
    unsigned int           missSBTIndex = rayType;
    unsigned int&          p0,
    unsigned int&          p1 );
float3 org = make_float3(0,0,0);
  float3 dir = make_float3(0,1,0);
  const uint32_t prd0 = 0;
  const uint32_t prd1 = 0;
  optixTrace(rg.world,
             org,dir,1e-3f,1e+10f,0.f,0x0,
             rayType,numRayTypes,rayType,prd0,prd1);
  
  const int fbOfs = pixelID.x+rg.fbSize.x*pixelID.y;
  int pattern = (pixelID.x / 8) ^ (pixelID.y/8);
  rg.fbPtr[fbOfs]
    = optix::make_rgba((pattern&1) ? rg.color1 : rg.color0);
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  printf("hit!!!\n");
  // if (optix::getLaunchIndex() == optix::vec2i(0))
  //   printf("Hello OptiX From your First RayGen Program\n");
}

OPTIX_MISS_PROGRAM(defaultRayType)()
{
  printf("miss!!!\n");
  // if (optix::getLaunchIndex() == optix::vec2i(0))
  //   printf("Hello OptiX From your First RayGen Program\n");
}

