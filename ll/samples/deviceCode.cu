#include "hip/hip_runtime.h"
// ======================================================================== //
// Copyright 2019 Ingo Wald                                                 //
//                                                                          //
// Licensed under the Apache License, Version 2.0 (the "License");          //
// you may not use this file except in compliance with the License.         //
// You may obtain a copy of the License at                                  //
//                                                                          //
//     http://www.apache.org/licenses/LICENSE-2.0                           //
//                                                                          //
// Unless required by applicable law or agreed to in writing, software      //
// distributed under the License is distributed on an "AS IS" BASIS,        //
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. //
// See the License for the specific language governing permissions and      //
// limitations under the License.                                           //
// ======================================================================== //

#include "deviceCode.h"

#define OPTIX_RAYGEN_PROGRAM(programName) \
  extern "C" __global__ \
  void __raygen__##programName

#define OPTIX_CLOSEST_HIT_PROGRAM(programName) \
  extern "C" __global__ \
  void __closesthit__##programName

#define OPTIX_MISS_PROGRAM(programName) \
  extern "C" __global__ \
  void __miss__##programName

OPTIX_RAYGEN_PROGRAM(simpleRayGen)()
{
  // RayGenData &rgData = *(RayGenData*)optix::getProgramDataPointer();
  const RayGenData &rg = optix::getProgramData<RayGenData>();
  const vec2i pixelID = optix::getLaunchIndex();
  if (pixelID == optix::vec2i(0)) {
    printf("%sHello OptiX From your First RayGen Program (on device %i/%i)%s\n",
           GDT_TERMINAL_LIGHT_RED,
           rg.deviceIndex,
           rg.deviceCount,
           GDT_TERMINAL_DEFAULT);
  }
  if (pixelID.x >= rg.fbSize.x) return;
  if (pixelID.y >= rg.fbSize.y) return;
  
  const int fbOfs = pixelID.x+rg.fbSize.x*pixelID.y;
  rg.fbPtr[fbOfs]
    = (fbOfs%2)
    ? rg.color1
    : rg.color0;
}

OPTIX_CLOSEST_HIT_PROGRAM(TriangleMesh)()
{
  // if (optix::getLaunchIndex() == optix::vec2i(0))
  //   printf("Hello OptiX From your First RayGen Program\n");
}

OPTIX_MISS_PROGRAM(defaultRayType)()
{
  // if (optix::getLaunchIndex() == optix::vec2i(0))
  //   printf("Hello OptiX From your First RayGen Program\n");
}

